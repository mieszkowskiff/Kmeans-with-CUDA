
#include <hip/hip_runtime.h>
#include <math.h>
#include <hiprand/hiprand_kernel.h>
#include <iostream>
#include <ctime>

#define n_DEFINED 2

#define CUDA_CHECK(cudaStatus)                                      \
    if(cudaStatus != hipSuccess)                                   \
        std::cout << hipGetErrorString(cudaStatus) << std::endl;   \


__device__ float standardNormal(hiprandState* state) {
    float u1 = hiprand_uniform(state);
    float u2 = hiprand_uniform(state);
    return sqrt(-2.0 * log(u1)) * cos(2.0 * M_PI * u2);
}


__global__ void generate_data_kernel(
    long N, 
    int n, 
    int n_classes, 
    float* mi, 
    float* sigma, 
    float* data, 
    int* labels, 
    hiprandState* states,
    unsigned long long seed) {
    // this function generates random data for the classification problem
    // N - number of points for each class
    // n - number of features
    // n_classes - number of classes
    // mi - pointer to the mi array
    // sigma - pointer to the sigma array
    // data - pointer to the data array
    // labels - pointer to the labels array

    

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N * n_classes) {
        return;
    }

    hiprand_init(idx + seed, idx, 0, &states[idx]);

    int class_idx = idx / N;
    
    labels[idx] = class_idx;
    

    // generate random data
    float sample[n_DEFINED];
    for (int i = 0; i < n; i++) {
        sample[i] = standardNormal(&states[idx]);
    }

    // transform sample to desired distribution
    float transformed_sample[n_DEFINED];
    for (int i = 0; i < n; i++) {
        transformed_sample[i] = mi[class_idx + i * n_classes];
        for (int j = 0; j <= i; j++) {
            transformed_sample[i] += sigma[class_idx + n_classes * (i * (i + 1) / 2 + j)] * sample[j];
        }
        data[idx + i * N * n_classes] = transformed_sample[i];
    }
}




void generate_data(long N, int n, int n_classes, float *data, int *labels, float spread, float skewness) {
    // this function generates random data for the classification problem
    // N - number of points for each class
    // n - number of features
    // n_classes - number of classes
    // data - pointer to the data array
    // labels - pointer to the labels array


    std::srand(static_cast<unsigned>(std::time(0)));
    // generate random mi and sigma
    float mi[n_classes * n];
    for (int i = 0; i < n_classes * n; i++) {
        mi[i] = ((float)rand() / RAND_MAX - 0.5) * spread;
    }


    // we represent only the lower triangular part of the matrix
    float sigma[n_classes * n * (n + 1) / 2];
    for(int k = 0; k < n_classes; k++) {
        for (int i = 0; i < n; i++) {
            for (int j = 0; j < i; j++) {
                // non diagonal elements
                sigma[k + n_classes * (i * (i + 1) / 2 + j)] = ((float)rand() / RAND_MAX) * skewness;
            }
            // diagonal elements
            sigma[k + n_classes * (i * (i + 1) / 2 + i)] = 0.2 + ((float)rand() / RAND_MAX) * 0.1;
        }
    }



    // allocate memory on the device
    int bytes_for_data = N * n_classes * n * sizeof(float);
    float* d_data;

    int bytes_for_labels = N * n_classes * sizeof(int);
    int* d_labels;

    CUDA_CHECK(hipMalloc((void**)&d_data, bytes_for_data));
    CUDA_CHECK(hipMalloc((void**)&d_labels, bytes_for_labels));

    

    // allocate memory for mi and sigma on the device
    float* d_mi;
    float* d_sigma;

    CUDA_CHECK(hipMalloc((void**)&d_mi, n_classes * n * sizeof(float)));
    CUDA_CHECK(hipMalloc((void**)&d_sigma, n_classes * n * (n + 1) / 2 * sizeof(float)));

    // transfer mi and sigma to the device
    CUDA_CHECK(hipMemcpy(d_mi, mi, n_classes * n * sizeof(float), hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_sigma, sigma, n_classes * n * (n + 1) / 2 * sizeof(float), hipMemcpyHostToDevice));

    int threads = 256;
    int blocks = N * n_classes / threads + 1;


    hiprandState* d_state;
    CUDA_CHECK(hipMalloc((void**)&d_state, N * n_classes * sizeof(hiprandState)));
    // generate data and labels

    unsigned long long seed = static_cast<unsigned long long>(time(NULL));
    printf("seed: %llu\n", seed);

    generate_data_kernel<<<blocks, threads>>>(N, n, n_classes, d_mi, d_sigma, d_data, d_labels, d_state, seed);

    hipDeviceSynchronize();
    // copy the data and labels from the device to the host
    CUDA_CHECK(hipMemcpy(data, d_data, bytes_for_data, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipMemcpy(labels, d_labels, bytes_for_labels, hipMemcpyDeviceToHost));


    CUDA_CHECK(hipFree(d_data));
    CUDA_CHECK(hipFree(d_labels));
    CUDA_CHECK(hipFree(d_mi));
    CUDA_CHECK(hipFree(d_sigma));
    CUDA_CHECK(hipFree(d_state));
}
