#include "hip/hip_runtime.h"
#include "helper.h"
#include <iostream>

#define CUDA_CHECK(cudaStatus)                                      \
    if(cudaStatus != hipSuccess)                                   \
        std::cout << hipGetErrorString(cudaStatus) << std::endl;   \

__global__ k_means_step(int N, int n, float* data, float* old_centroids, float* new_centroids) {
    // this function performs one step of the k-means algorithm
    // N - number of data points
    // n - number of features
    // data - pointer to the data array
    // old_centroids - pointer to the old centroids array
    // new_centroids - pointer to the new centroids array

    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= N) {
        return;
    }


    // update the new centroid
    for (int i = 0; i < n; i++) {
        atomicAdd(&new_centroids[closest_centroid * n + i], data[idx * n + i]);
    }
}
void k_means(int N, int n, float* data, float k, float* centroids, int iterations) {
    
    // Allocate memory for the old centroids
    float* d_centroids1;
    CUDA_CHECK(hipMalloc(&d_centroids1, k * n * sizeof(float)));
    CUDA_CHECK(hipMemcpy(d_centroids1, centroids, k * n * sizeof(float), hipMemcpyHostToDevice));


    // Allocate memory for the new centroids
    float* d_centroids2;
    CUDA_CHECK(hipMalloc(&d_centroids2, k * n * sizeof(float)));


    // Allocate memory for the data
    float* d_data;
    CUDA_CHECK(hipMalloc(&d_data, N * n * sizeof(float)));

    // Transfer the data to the device
    CUDA_CHECK(hipMemcpy(d_data, data, N * n * sizeof(float), hipMemcpyHostToDevice));





    // depending on the iteration number, the score will be at centroid 1 or 2
    if (iterations % 2 == 0) {
        CUDA_CHECK(hipMemcpy(centroids, d_centroids1, k * n * sizeof(float), hipMemcpyDeviceToHost));
    } else {
        CUDA_CHECK(hipMemcpy(centroids, d_centroids2, k * n * sizeof(float), hipMemcpyDeviceToHost));
    }


    CUDA_CHECK(hipFree(d_data));
    CUDA_CHECK(hipFree(d_centroids1));
    CUDA_CHECK(hipFree(d_centroids2));


}